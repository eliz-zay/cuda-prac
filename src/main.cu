#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <map>
#include <string>
#include <vector>
#include <stdbool.h>
#include <tuple>
#include <unistd.h>
#include <sys/types.h>
#include <dirent.h>
#include <stdio.h>

#include "../external/lodepng/lodepng.cpp"

#define sqr(a) ((a)*(a))

using namespace std;

vector<string> listdir(const char *name, int indent) {
    DIR *dir;
    struct dirent *entry;
    vector<string> files;

    if (!(dir = opendir(name)))
        return files;

    while ((entry = readdir(dir)) != NULL) {
        if (entry->d_type == DT_DIR) {
            char path[1024];
            if (strcmp(entry->d_name, ".") == 0 || strcmp(entry->d_name, "..") == 0)
                continue;
            snprintf(path, sizeof(path), "%s/%s", name, entry->d_name);
            files.push_back(entry->d_name);
            listdir(path, indent + 2);
        } else {
            files.push_back(entry->d_name);
        }
    }
    closedir(dir);

    return files;
}

const map<string, pair<char, float* > > kernels {
    {
        "box-blur",
        {
            3,
            new float[9] {
                1.0/9, 1.0/9, 1.0/9,
                1.0/9, 1.0/9, 1.0/9,
                1.0/9, 1.0/9, 1.0/9
            }
        }
    },
    {
        "gaussian-blur-3",
        {
            3,
            new float[9] {
                1.0/16, 2.0/16, 1.0/16,
                2.0/16, 4.0/16, 2.0/16,
                1.0/16, 2.0/16, 1.0/16
            }
        }
    },
    {
        "gaussian-blur-5",
        {
            5,
            new float[25] {
                1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256,
                4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
                6.0/256, 24.0/256, 36.0/256, 24.0/256, 6.0/256,
                4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
                1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256
            }
        }
    },
    {
        "edge-detect",
        {
            3,
            new float[9] {
                -1, -1, -1,
                -1,  8, -1,
                -1, -1, -1
            }
        }
    },
    {
        "emboss",
        {
            3,
            new float[9] {
                -2, -1, 0,
                -1,  1, 1,
                 0,  1, 2
            }
        }
    }
};

int maxBlockSize;
int maxBlockDimX;
int maxBlockDimY;
int maxGridDimX;
int maxGridDimY;

void getError(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA error - %s\n", hipGetErrorString(err));
    }
}

__global__ void apply_kernel_device(
    unsigned char* input_image,
    unsigned char* output_image,
    int width,
    int height,
    float* kernel,
    char kernel_dim
) {
    const unsigned int linearX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;

    if (linearX >= width || linearY >= height) {
        return;
    }

    float r;
    float g;
    float b;

    /**
     * Case when kernel dimension is 3. In this case process all pixels but first edge
     */
    if (kernel_dim == 3 && linearX > 0 && linearX < width - 1 && linearY > 0 && linearY < height - 1) {
        r = 0;
        g = 0;
        b = 0;

        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                r += input_image[3 * ((linearY + i) * width + (linearX + j))] * kernel[3 * (i + 1) + j + 1];
                g += input_image[3 * ((linearY + i) * width + (linearX + j)) + 1] * kernel[3 * (i + 1) + j + 1];
                b += input_image[3 * ((linearY + i) * width + (linearX + j)) + 2] * kernel[3 * (i + 1) + j + 1];
            }
        }

        output_image[3 * (linearY * width + linearX)] = ceil(r);
        output_image[3 * (linearY * width + linearX) + 1] = ceil(g);
        output_image[3 * (linearY * width + linearX) + 2] = ceil(b);
    }
    /**
     * Case when kernel dimension is 5. In this case process all pixels but first two edges
     */
    else if ((kernel_dim == 5 && linearX > 1 && linearX < width - 2 && linearY > 2 && linearY < height - 2)) {
        r = 0;
        g = 0;
        b = 0;

        for (int i = -2; i < 3; i++) {
            for (int j = -2; j < 3; j++) {
                r += input_image[3 * ((linearY + i) * width + (linearX + j))] * kernel[3 * (i + 1) + j + 1];
                g += input_image[3 * ((linearY + i) * width + (linearX + j)) + 1] * kernel[3 * (i + 1) + j + 1];
                b += input_image[3 * ((linearY + i) * width + (linearX + j)) + 2] * kernel[3 * (i + 1) + j + 1];
            }
        }

        output_image[3 * (linearY * width + linearX)] = ceil(r);
        output_image[3 * (linearY * width + linearX) + 1] = ceil(g);
        output_image[3 * (linearY * width + linearX) + 2] = ceil(b);
    }
    /**
     * Case when pixel is on the edge
     */
    else {
        output_image[3 * (linearY * width + linearX)] = input_image[3 * (linearY * width + linearX)];
        output_image[3 * (linearY * width + linearX) + 1] = input_image[3 * (linearY * width + linearX) + 1];
        output_image[3 * (linearY * width + linearX) + 2] = input_image[3 * (linearY * width + linearX) + 2];
    }
}

void apply_kernel(unsigned char* input_image, unsigned char* output_image, int width, int height, string filter) {
    unsigned char* dev_input;
    unsigned char* dev_output;
    float* dev_kernel;

    float ms_outer = 0;
    float ms_inner = 0;
    hipEvent_t start_outer;
    hipEvent_t stop_outer;
    hipEvent_t start_inner;
    hipEvent_t stop_inner;
    hipEventCreate(&start_outer);
    hipEventCreate(&stop_outer);
    hipEventCreate(&start_inner);
    hipEventCreate(&stop_inner);

    hipEventRecord(start_outer);
    hipEventSynchronize(start_outer);

    getError(hipMalloc((void **)&dev_input, 3 * width * height * sizeof(unsigned char)));
    getError(hipMemcpy(dev_input, input_image, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    getError(hipMalloc((void **)&dev_kernel, sqr(kernels.at(filter).first) * sizeof(float)));
    getError(hipMemcpy(dev_kernel, kernels.at(filter).second, sqr(kernels.at(filter).first) * sizeof(float), hipMemcpyHostToDevice));

    getError(hipMalloc((void **)&dev_output, 3 * width * height * sizeof(unsigned char)));

    int blockDim = min(static_cast<int>(floor(sqrt(maxBlockSize))), min(maxBlockDimX, maxBlockDimY));
    int gridDimX = ceil(1.0 * width / blockDim);
    int gridDimY = ceil(1.0 * height / blockDim);

    if (gridDimX > maxGridDimX || gridDimY > maxGridDimY) {
        throw runtime_error("Too big image");
    }

    printf("Device params: block size %d, grid x-dim %d, grid y-dim %d\n", blockDim, gridDimX, gridDimY);

    dim3 blockDims(blockDim, blockDim, 1);
    dim3 gridDims(gridDimX, gridDimY, 1);

    hipEventRecord(start_inner);
    hipEventSynchronize(start_inner);

    apply_kernel_device<<<gridDims, blockDims>>>(dev_input, dev_output, width, height, dev_kernel, kernels.at(filter).first);

    hipEventRecord(stop_inner);
    hipEventSynchronize(stop_inner);
    hipEventElapsedTime(&ms_inner, start_inner, stop_inner);

    getError(hipMemcpy(output_image, dev_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));
    getError(hipFree(dev_kernel));

    hipEventRecord(stop_outer);
    hipEventSynchronize(stop_outer);
    hipEventElapsedTime(&ms_outer, start_outer, stop_outer);

    printf("GPU calculation time: %g ms\n", ms_inner);
    printf("GPU calculation + transport time: %g ms\n", ms_outer);
}

void loadCudaSettings() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    maxBlockSize = prop.maxThreadsPerBlock;
    maxBlockDimX = prop.maxThreadsDim[0];
    maxBlockDimY = prop.maxThreadsDim[1];
    maxGridDimX = prop.maxGridSize[0];
    maxGridDimY = prop.maxGridSize[1];

    printf("CUDA block max size - %d\n", prop.maxThreadsPerBlock);
    printf("CUDA block max dimensions - %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("CUDA grid max dimensions - %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void processImage(char* input_file, char* output_file, string filter) {
    vector<unsigned char> in_image;
    unsigned int width, height;

    unsigned error = lodepng::decode(in_image, width, height, input_file);
    if (error) {
        cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;
    }

    unsigned char *input_image = new unsigned char[(in_image.size() * 3) / 4];
    unsigned char *output_image = new unsigned char[(in_image.size() * 3) / 4];
    int inp_iterator = 0;
    for (int i = 0; i < in_image.size(); ++i) {
        if ((i + 1) % 4 != 0) {
            input_image[inp_iterator] = in_image.at(i);
            output_image[inp_iterator] = 255;
            inp_iterator++;
        }
    }

    printf("Image size - %dx%d\n", width, height);

    apply_kernel(input_image, output_image, width, height, filter);

    int out_iterator = 0;
    vector<unsigned char> out_image(in_image.size());
    for (int i = 0; i < width * height * 3; ++i) {
        out_image[out_iterator] = output_image[i];
        out_iterator++;
        if ((i + 1) % 3 == 0) {
            out_image[out_iterator] = 255;
            out_iterator++;
        }
    }

    error = lodepng::encode(output_file, out_image, width, height);

    if (error) {
        printf("Encoder error: %s\n", lodepng_error_text(error));
    }

    delete[] input_image;
    delete[] output_image;
}

void parseArgs(int argc, char** argv, char** filter, char** imgType) {
    if (argc != 3) {
        printf("2 arguments required");
        exit(0);
    }

    *filter = argv[1];
    *imgType = argv[2];
}

int main(int argc, char** argv) {
    vector<pair<char*, char*> > images;
    char* kernel, *imgType;

    parseArgs(argc, argv, &kernel, &imgType);

    if (!strcmp(imgType,"big")) {
        images.push_back({ "in/big.png", "out/big.png" });

    } else if (!strcmp(imgType,"small")) {
        vector<string> files = listdir("in/small/", 0);
        for (string name: files) {
            string strIn = "in/small/" + name;
            string strOut = "out/" + name;
            char *in = new char[strIn.length() + 1];
            char *out = new char[strOut.length() + 1];
            strcpy(in, strIn.c_str());
            strcpy(out, strOut.c_str());
            images.push_back({ in, out });
        }
    } else {
        cout << "Invalid arguments" << endl;
        return 0;
    }

    loadCudaSettings();

    string str(kernel);

    for (int i = 0; i < images.size(); i++) {
        printf("Started processing image %s\n", images[i].first);
        processImage(images[i].first, images[i].second, str);
        printf("Finised. Output was written to %s\n", images[i].second);
    }
    
    return 0;
}